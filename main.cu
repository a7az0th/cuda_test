
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(float *a, float *out, int n) {
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if (i >= n) {
        return;
    }
    out[i] = a[i] + a[i];
}

#define checkErr(X) \
{ \
  hipError_t err = X;\
  if (err != hipSuccess) { \
    printf("%s[%d] CUDA Error %d\n", __func__, __LINE__, err); \
    exit(err); \
  } \
}

int main() {
    const size_t N = 10;	
    float a[N];
    float *d_a = nullptr;
    float *d_b = nullptr;

    for (int i = 0; i < N; i++) {
        a[i] = i;
    }

    // Allocate device memory for a
    checkErr(hipMalloc((void**)&d_a, sizeof(float) * N));
    checkErr(hipMalloc((void**)&d_b, sizeof(float) * N));

    // Transfer data from host to device memory
    checkErr(hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice));

    vector_add<<<1, N>>>(d_a, d_b, N);

    checkErr(hipMemcpy(a, d_b, sizeof(float) * N, hipMemcpyDeviceToHost));

    // Cleanup after kernel execution
    checkErr(hipFree(d_a));
    checkErr(hipFree(d_b));

    checkErr(hipDeviceSynchronize());

    for (int i = 0; i < N; i++) {
        printf("%.2f, ", a[i]);
    }
    printf("\n");
    return 0;
}
